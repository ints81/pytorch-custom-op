#include "hip/hip_runtime.h"
#include <torch/library.h>
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Operators.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void add_kernel(int numel, const float* a, const float* b, float* c) {
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (gidx < numel) {
        c[gidx] = a[gidx] + b[gidx];
    }
}

at::Tensor add_cuda(const at::Tensor& a, const at::Tensor& b) {
    TORCH_CHECK(a.device().type() == at::DeviceType::CUDA);
    TORCH_CHECK(b.device().type() == at::DeviceType::CUDA);
    TORCH_CHECK(a.dtype() == at::kFloat);
    TORCH_CHECK(b.dtype() == at::kFloat);
    TORCH_CHECK(a.sizes() == b.sizes());

    at::Tensor a_contig = a.contiguous();
    at::Tensor b_contig = b.contiguous();
    at::Tensor c = at::empty_like(a_contig);

    hipStream_t stream = c10::cuda::getCurrentCUDAStream();
    int numel = a_contig.numel();
    const float* a_ptr = a_contig.data_ptr<float>();
    const float* b_ptr = b_contig.data_ptr<float>();
    float* c_ptr = c.data_ptr<float>();
    add_kernel<<<(numel + 255) / 256, 256, 0, stream>>>(numel, a_ptr, b_ptr, c_ptr);

    return c;
}

// 반드시 m.impl의 첫 argument의 이름은 위에서 선언한 다른 function의 이름과 달라야 함
TORCH_LIBRARY_IMPL(pytorch_custom_op, CUDA, m) {
    m.impl("myadd", &add_cuda);
}